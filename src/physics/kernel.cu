#include "hip/hip_runtime.h"
#include "kernel.h"
#include <thrust/device_ptr.h>
#include <thrust/count.h>
#include <thrust/copy.h>

#define BUFFER_OFFSET(i) ((char *)NULL + (i))

/*****************
* Configuration *
*****************/

#define blockSize 128
//#define DEBUG

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

// kernel size
int numOfModules;
int numOfEdges;
dim3 threadsPerBlock(blockSize);

// buffers to hold in our graph data
Node* dev_nodes;
Edge* dev_edges;
Module* dev_modules;
ModuleEdge* dev_moduleEdges;
GridCell* dev_gridCells;

// indices of tree buffers (needed for culling)
int* dev_moduleIndices;
int* dev_temp_moduleIndices;

// grid cell module adjacency
GridModuleAdj* dev_gridModuleAdjs;

// Grid Kernel Launch params
const dim3 M_in(M_IX, M_IY, M_IZ);

// TODO add rest of grid params
float* dev_temp;
float* dev_oldtemp;
float3* dev_vel;
float3* dev_oldvel;
float* dev_pressure;
float3* dev_ccvel;
float3* dev_vorticity;
float* dev_smokedensity;
float* dev_oldsmokedensity;
float* dev_smokeRadiance;
float* dev_deltaM;

Terrain* m_terrain;

/******************
* initSimulation *
******************/

void Simulation::initSimulation(Terrain* terrain, int3 gridCount)
{
    m_terrain = terrain;
    numOfModules = terrain->modules.size();
    numOfEdges = terrain->edges.size();

    int numOfNodes = terrain->nodes.size();
    int numOfGrids = gridCount.x * gridCount.y * gridCount.z;

    // Allocate buffers for the modules
    HANDLE_ERROR(hipMalloc((void**)&dev_nodes, numOfNodes * sizeof(Node)));
    HANDLE_ERROR(hipMemcpy(dev_nodes, terrain->nodes.data(), numOfNodes * sizeof(Node), hipMemcpyHostToDevice));

    HANDLE_ERROR(hipMalloc((void**)&dev_edges, numOfEdges * sizeof(Edge)));
    HANDLE_ERROR(hipMemcpy(dev_edges, terrain->edges.data(), numOfEdges * sizeof(Edge), hipMemcpyHostToDevice));

    HANDLE_ERROR(hipMalloc((void**)&dev_modules, numOfModules * sizeof(Module)));
    HANDLE_ERROR(hipMemcpy(dev_modules, terrain->modules.data(), numOfModules * sizeof(Module), hipMemcpyHostToDevice));

    HANDLE_ERROR(hipMalloc((void**)&dev_moduleEdges, terrain->moduleEdges.size() * sizeof(ModuleEdge)));
    HANDLE_ERROR(hipMemcpy(dev_moduleEdges, terrain->moduleEdges.data(), terrain->moduleEdges.size() * sizeof(ModuleEdge), hipMemcpyHostToDevice));

    HANDLE_ERROR(hipMalloc((void**)&dev_moduleIndices, numOfModules * sizeof(Module)));
    HANDLE_ERROR(hipMalloc((void**)&dev_temp_moduleIndices, numOfModules * sizeof(Module)));

    // Allocate grid buffers
    HANDLE_ERROR(hipMalloc((void**)&dev_temp, numOfGrids * sizeof(float)));

    HANDLE_ERROR(hipMalloc((void**)&dev_oldtemp, numOfGrids * sizeof(float)));

    HANDLE_ERROR(hipMalloc((void**)&dev_vel, numOfGrids * sizeof(float3)));

    HANDLE_ERROR(hipMalloc((void**)&dev_oldvel, numOfGrids * sizeof(float3)));

    HANDLE_ERROR(hipMalloc((void**)&dev_pressure, numOfGrids * sizeof(float)));

    HANDLE_ERROR(hipMalloc((void**)&dev_ccvel, numOfGrids * sizeof(float3)));
    HANDLE_ERROR(hipMemset(dev_ccvel, 0, numOfGrids * sizeof(float3)));

    HANDLE_ERROR(hipMalloc((void**)&dev_vorticity, numOfGrids * sizeof(float3)));
    HANDLE_ERROR(hipMemset(dev_vorticity, 0, numOfGrids * sizeof(float3)));

    HANDLE_ERROR(hipMalloc((void**)&dev_smokedensity, numOfGrids * sizeof(float)));

    HANDLE_ERROR(hipMalloc((void**)&dev_oldsmokedensity, numOfGrids * sizeof(float)));

    HANDLE_ERROR(hipMalloc((void**)&dev_smokeRadiance, numOfGrids * sizeof(float)));

    HANDLE_ERROR(hipMalloc((void**)&dev_deltaM, numOfGrids * sizeof(float)));

    HANDLE_ERROR(hipMalloc((void**)&dev_gridModuleAdjs, terrain->gridModuleAdjs.size() * sizeof(GridModuleAdj)));
    hipMemcpy(dev_gridModuleAdjs, terrain->gridModuleAdjs.data(), terrain->gridModuleAdjs.size() * sizeof(GridModuleAdj), hipMemcpyHostToDevice);

    HANDLE_ERROR(hipMalloc((void**)&dev_gridCells, terrain->gridCells.size() * sizeof(GridCell)));
    hipMemcpy(dev_gridCells, terrain->gridCells.data(), terrain->gridCells.size() * sizeof(GridCell), hipMemcpyHostToDevice);

    initGridBuffers(gridCount, dev_temp, dev_oldtemp, dev_vel, dev_oldvel, dev_smokedensity, dev_oldsmokedensity, dev_pressure, M_in);

    dim3 modules_fullBlocksPerGrid((numOfModules + blockSize - 1) / blockSize);
    
    kernInitModules << <modules_fullBlocksPerGrid, blockSize >> > (numOfModules, dev_nodes, dev_edges, dev_modules);

    kernInitIndices << <modules_fullBlocksPerGrid, blockSize >> > (numOfModules, dev_moduleIndices);

    hipDeviceSynchronize();
}

/******************
* stepSimulation *
******************/
struct is_negative
{
    __host__ __device__
        bool operator()(int x)
    {
        return x < 0;
    }
};

struct is_nonnegative
{
    __host__ __device__
        bool operator()(int x)
    {
        return x >= 0;
    }
};

void Simulation::stepSimulation(float dt, int3 gridCount, float3 gridSize, float sideLength, float* d_out)
{
    dim3 fullBlocksPerGrid((numOfModules + blockSize - 1) / blockSize);

    // For each module in the forest
    // - Update mass
    // - Perform radii update
    // - Update temperature
    // - Update released water content
    kernModuleCombustion << <fullBlocksPerGrid, blockSize >> > (dt, numOfModules, dev_moduleIndices, gridCount, sideLength, dev_nodes, dev_edges, dev_modules, dev_moduleEdges, dev_oldtemp);

    // For each grid point x in grid space
    // - update mass and water content
    // TODO: implement
    const dim3 gridDim(blocksNeeded(gridCount.x, M_IX), blocksNeeded(gridCount.y, M_IY), blocksNeeded(gridCount.z, M_IZ));
    
    kernComputeChangeInMass<<<gridDim, M_in>>>(gridCount, dev_modules, dev_gridCells, dev_gridModuleAdjs, dev_deltaM);
    HANDLE_ERROR(hipPeekAtLastError()); HANDLE_ERROR(hipDeviceSynchronize());

    // Update air temperature
    // update drag forces (wind)
    // update smoke density (qs), water vapor (qv), condensed water (qc),
    // and rain (qc)
    float* dev_lap;
    float3* dev_alpha_m;
    float3 externalForce = { 0.f, 0.f, 0.f };

    //float* h_deltaM = (float*)malloc(sizeof(float) * 24 * 8 * 24);
    //hipMemcpy(h_deltaM, dev_deltaM, sizeof(float) * 24 * 8 * 24, hipMemcpyDeviceToHost);
    //int num = 0;
    //for (int i = 0; i < 24 * 8 * 24; i++) {
    //    //if (h_temp[i] > 50.f) {
    //    //    printf("d_temp[%d] = %f\n", i, h_temp[i]);
    //    //}
    //    if (h_deltaM[i] != 0.f) {
    //        printf("d_deltaM[%d] = %f\n", i, h_deltaM[i]);
    //        num++;
    //    }
    //}
    //printf("%d\n", num);
    //free(h_deltaM);


    HANDLE_ERROR(hipMalloc(&dev_lap, gridCount.x * gridCount.y * gridCount.z * sizeof(float)));
    HANDLE_ERROR(hipMalloc(&dev_alpha_m, gridCount.x * gridCount.y * gridCount.z * sizeof(float3)));

    computeVorticity << <gridDim, M_in >> > (gridCount, sideLength, dev_vorticity, dev_oldvel, dev_ccvel);
    HANDLE_ERROR(hipPeekAtLastError()); HANDLE_ERROR(hipDeviceSynchronize());

    velocityKernel << <gridDim, M_in >> > (gridCount, gridSize, sideLength, dev_oldtemp, dev_vel, dev_oldvel, dev_alpha_m, dev_oldsmokedensity, dev_vorticity, externalForce);
    HANDLE_ERROR(hipPeekAtLastError()); HANDLE_ERROR(hipDeviceSynchronize());

    // Pressure Solve
    forceIncompressibility(gridCount, sideLength, dev_vel, dev_pressure);

    tempAdvectionKernel << <gridDim, M_in >> > (gridCount, gridSize, sideLength, dev_temp, dev_oldtemp, dev_vel, dev_alpha_m, dev_lap, dev_deltaM);
    HANDLE_ERROR(hipPeekAtLastError()); HANDLE_ERROR(hipDeviceSynchronize());

    //float* h_temp = (float*)malloc(sizeof(float) * 24 * 8 * 24);
    //hipMemcpy(h_temp, dev_temp, sizeof(float) * 24 * 8 * 24, hipMemcpyDeviceToHost);
    //int num = 0;
    //for (int i = 0; i < 24 * 8 * 24; i++) {
    //    //if (h_temp[i] > 50.f) {
    //    //    printf("d_temp[%d] = %f\n", i, h_temp[i]);
    //    //}
    //    if (h_temp[i] > 30.f) {
    //        num++;
    //    }
    //}
    ////printf("%d\n", num);
    //free(h_temp);

    smokeUpdateKernel << <gridDim, M_in >> > (gridCount, gridSize, sideLength, dev_oldtemp, dev_vel, dev_alpha_m, dev_smokedensity, 
        dev_oldsmokedensity, dev_deltaM);

    //printf("FINISH ITER\n");

    smokeRender(gridCount, gridSize, sideLength, gridDim, M_in, d_out, dev_smokedensity, dev_smokeRadiance, dev_oldtemp);

    HANDLE_ERROR(hipPeekAtLastError());
    HANDLE_ERROR(hipDeviceSynchronize());

    HANDLE_ERROR(hipFree(dev_alpha_m));
    HANDLE_ERROR(hipFree(dev_lap));

    // Ping-pong buffers
    std::swap(dev_temp, dev_oldtemp);
    std::swap(dev_vel, dev_oldvel);
    std::swap(dev_smokedensity, dev_oldsmokedensity);

    // For each module in the forest
    // cull modules (and their children) that have zero mass
    
    kernCullModules1 << <fullBlocksPerGrid, blockSize >> > (numOfModules, dev_moduleIndices, dev_modules, dev_moduleEdges, dev_nodes, dev_edges);
    HANDLE_ERROR(hipPeekAtLastError());
    HANDLE_ERROR(hipDeviceSynchronize());
    kernCullModules2 << <fullBlocksPerGrid, blockSize >> > (numOfModules, dev_moduleIndices, dev_modules, dev_moduleEdges, dev_nodes, dev_edges);
    HANDLE_ERROR(hipPeekAtLastError());
    HANDLE_ERROR(hipDeviceSynchronize());

    // stream compaction
    thrust::device_ptr<int> thrust_indices = 
        thrust::device_pointer_cast(dev_moduleIndices);
    thrust::device_ptr<int> thrust_temp =
        thrust::device_pointer_cast(dev_temp_moduleIndices);

    int numCulled = thrust::count_if(thrust_indices, thrust_indices + numOfModules * sizeof(int), is_negative());
    thrust::copy_if(thrust_indices, thrust_indices + numOfModules * sizeof(int), thrust_temp, is_nonnegative());

    // update data
    numOfModules -= numCulled;
    dev_moduleIndices = thrust::raw_pointer_cast(thrust_indices);
    dev_temp_moduleIndices = thrust::raw_pointer_cast(thrust_temp);

    // ping pong buffers
    std::swap(dev_moduleIndices, dev_temp_moduleIndices);
    
    hipDeviceSynchronize();
}

/****************
* endSimulation *
*****************/
void Simulation::endSimulation()
{
    // Send back to host to check
    HANDLE_ERROR(hipMemcpy(m_terrain->nodes.data(), dev_nodes, m_terrain->nodes.size() * sizeof(Node), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(m_terrain->edges.data(), dev_edges, m_terrain->edges.size() * sizeof(Edge), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(m_terrain->modules.data(), dev_modules, m_terrain->modules.size() * sizeof(Module), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(m_terrain->moduleEdges.data(), dev_moduleEdges, m_terrain->moduleEdges.size() * sizeof(ModuleEdge), hipMemcpyDeviceToHost));

    hipFree(dev_gridModuleAdjs);
    hipFree(dev_modules);
    hipFree(dev_edges);
    hipFree(dev_nodes);

    hipDeviceSynchronize();
}

/********************
* copyBranchesToVBO *
*********************/

// TODO: need to cull edges
__global__ void kernUpdateVBOBranches(int N, float* vbo, Node* nodes, Edge* edges, Module* modules)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) return;

    Edge& edge = edges[index];

    if (!edge.culled)
    {
        Node& fromNode = nodes[edge.fromNode];
        Node& toNode = nodes[edge.toNode];

        vbo[11 * index + 0] = fromNode.position.x;
        vbo[11 * index + 1] = fromNode.position.y;
        vbo[11 * index + 2] = fromNode.position.z;
        vbo[11 * index + 3] = fromNode.radius;

        vbo[11 * index + 4] = toNode.position.x;
        vbo[11 * index + 5] = toNode.position.y;
        vbo[11 * index + 6] = toNode.position.z;
        vbo[11 * index + 7] = toNode.radius;

        vbo[11 * index + 8] = (fromNode.leaf && modules[edge.moduleInx].temperature < 300) ? 1.0f : -1.f;
        vbo[11 * index + 9] = (toNode.leaf && modules[edge.moduleInx].temperature < 300) ? 1.0f : -1.f;
        vbo[11 * index + 10] = modules[edge.moduleInx].temperature;
    }
    else
    {
        for (int i = 0; i < 11; i++)
        {
            vbo[11 * index + i] = 0.f;
        }
    }
}

void Simulation::copyBranchesToVBO(float* vbodptr_branches)
{
    // TODO: implement
    dim3 fullBlocksPerGrid((numOfEdges + blockSize - 1) / blockSize);
    kernUpdateVBOBranches << <fullBlocksPerGrid, blockSize >>> (numOfEdges, vbodptr_branches, dev_nodes, dev_edges, dev_modules);
    hipDeviceSynchronize();
}
