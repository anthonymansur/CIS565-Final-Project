#include <hip/hip_runtime.h>
#include "kernel.h"

/*****************
* Configuration *
*****************/

#define blockSize 128;



/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

// buffers to hold in our graph data
Node* dev_nodes;
Edge* dev_edges;
Module* dev_modules;

/******************
* initSimulation *
******************/
void Simulation::initSimulation(Terrain* terrain)
{
    // Allocate buffers for the modules
    hipMalloc((void**)&dev_nodes, terrain->nodes.size() * sizeof(Node));
    hipMemcpy(dev_nodes, terrain->nodes.data(), terrain->nodes.size(), hipMemcpyHostToDevice);

    hipMalloc((void**)&dev_edges, terrain->edges.size() * sizeof(Edge));
    hipMemcpy(dev_edges, terrain->edges.data(), terrain->edges.size(), hipMemcpyHostToDevice);

    hipMalloc((void**)&dev_modules, terrain->modules.size() * sizeof(Module));
    hipMemcpy(dev_modules, terrain->modules.data(), terrain->modules.size(), hipMemcpyHostToDevice);
    
    // TODO: check cuda error
}

/******************
* stepSimulation *
******************/
void Simulation::stepSimulation(float dt)
{
    // TODO: implement
}

/******************
* endSimulation *
******************/
void Simulation::endSimulation()
{
    hipFree(dev_modules);
    hipFree(dev_edges);
    hipFree(dev_nodes);
}