#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES // Keep above math.h import
#include <math.h> 
#include "module.h"

/************
* Constants *
*************/
// ---------------- TUNABLE ----------------
/**
 * @brief char contraction factor
 * @note range: [0.5, 1.0], units: 1
 * Tunable: inversely proportional to combustibility
 */
__device__ const float k = 0.75;

/**
 * @brief minimum value of c as a result of charring
 * @note range: [0.0, 1.0], units: 1
 * Tunable: directly proportional to combustibility
 */
__device__ const float c_min = 1/*0.5*/;

/**
 * @brief the rate of insulation due to charring
 * @note range: [50,250], units: 1 m^-1
 * Tunable: directly proportional to combustibility
 */
__device__ const float c_r = 250/*150*/;

/**
 * @brief Mass loss rate
 * @note range: [0.01, 10.0] x 10^-3
 * Tunable: directly proportional to combustibility
 */
__device__ const float delta_m = 10e-3;

/** Scalar for the adjacenet module temperature diffusion term */
__device__ float lap_constant = 0.15f; // TODO: TUNE

/**
 * @brief the ambient temperature of the modules
 * @note units: celsius
 * Tunable
 */
__device__ const float T_amb = 15.f; // TODO: move elsewhere?

/**
 * @brief Heat transfer coeff. for dry wood
 * @note range: [0.03, 0.1], units: 1 s^-1
 */ 
#define B_DRY 0.1                           // TODO: TUNE
__device__ const float b_dry = B_DRY; 

/** Used to determine if module has no more mass */
__device__ const float MASS_EPSILON = FLT_EPSILON; // TODO: update


/** Clamping values for change in mass and temperature of modules */
__device__ const float MAX_DELTA_M = 0.1f;// 0.0001;  // TODO: TUNE 
__device__ const float MAX_DELTA_T = 1000.f;//0.001; // TODO: TUNE

// ---------------- NOT TUNABLE ----------------
/**
 * @brief the temperatures used in combustion
 * @note min: 150, max: 450, units: celsius
 * Not tunable
 */
__device__ const float T0 = 150;
__device__ const float T1 = 450;

/**
 * @brief the saturation temperature of water
 * @note constant, units: celsius
 */
 __device__ const float T_sat = 100;

/**
 * @brief the maximum wind boost
 * @note range: XXX, units: 2 kg s^-1 m^-2
 */
__device__ const float n_max = 2;

/**
 * @brief the maximum wind velocity
 */
__device__ const float u_ref = 15;

/**
 * @brief used in the heat flux to water per unit area calculation
 * @note range: constant, units: 1 Wm^-2 celsius^-1
 * Not tunable
 */
__device__ const float c_bar = 0.1;

/**
 * @brief Specific heat capacity of a module
 * @note range: constant, units: 1 kJ celsius^-1 kg
 * Not tunable
 */
__device__ const float c_M = 2.5;

/**
 * @brief density of wood
 * @note units: 1 kg m^-3, deciduous = 660, conifer = 420, shrub = 300
 * Not tunable
 */
__device__ const float rho = 660; // WARNING: see note below.
// NOTE: center of mass calculations currently assumes this to be equal to 660.

 /**
 * @brief Temperature diffusion coeff. (module)
 * @note range: XXX, units: 1 m^2 s^-1
 * Not tunable
 */
__device__ float alpha_M = 0.75;

/**
 * @brief Heat transfer coeff. for wet wood 
 * @note range: 0.1 * b_dry, units: 1 s^-1
 * Not tunable
 */
__device__ const float b_wet = 0.1 * B_DRY;

/**
 * @brief ratio of water released to wood burned
 * @note range: 0.5362 kg water per kg of wood
 * Not tunable
 */
__device__ const float c_WM = 0.5362;

/*******************
* Device Functions *
********************/
__device__ float sigmoid(float x)
{
    return 3 * x * x - 2 * x * x * x;
}

__device__ float getArea(float r0, float r1, float l)
{
    return (float)M_PI * (r0 + r1) * sqrtf((r0 - r1) * (r0 - r1) + l * l);
}

__device__ float getVolume(float r0, float r1, float l)
{
    return (float)(M_PI / 3) * l * (r0 * r0 + r0 * r1 + r1 * r1);
}

__device__ float windSpeedFunction(float u)
{
    return (n_max - 1) * sigmoid(u / u_ref) + 1;
}

__device__ float computeReactionRate(float temp, float windSpeed)
{
    if (temp < T0)
        return 0;
    else if (temp > T1)
        return 1;
    else
        return windSpeedFunction(windSpeed) * sigmoid((temp - T0) / (T1 - T0));
}

__device__ float heightOfPyrolyzingFront(float H0, float A0, float mass)
{
    return sqrt(2 * (mass / rho) * (H0 / A0));
}

__device__ float charLayerThickness(float H0, float H)
{
    return k * (H0 - H);
}

__device__ float charLayerInsulation(float H_c)
{
    return c_min + (1 - c_min)*exp(-c_r * H_c);
}

__device__ float getFrontArea(float A0, float H0, float H)
{
    return A0 * H / H0;
}

__device__ float rateOfMassChange(float mass, float H0, float H, float A0, float temp, float frontArea, float windSpeed)
{
    float H_c = charLayerThickness(H0, H);
    float c = charLayerInsulation(H_c);
    float k = computeReactionRate(temp, windSpeed);

    // TODO: verify this is correct, as it's throwing nan
    return -delta_m * k * c * frontArea;
}

__device__ float radiiModuleConstant(Node* nodes, Edge* edges, Module& module)
{
    /** Replace code with what's commented for simplier solution */
    //Node& node = nodes[module.startNode];
    //return node.radius / sqrt((3 / (M_PI * rho)) * module.mass);

    float sum = 0;
    for (int i = module.startEdge; i <= module.lastEdge; i++)
    {
        // For every edge in the module, do the following:

        Edge* edge = &edges[i]; // will be updated
        float l = edge->length;
        float lambda = edge->radiiRatio;
        float prod = 1;

        // check to see if fromNode isn't the root node
        while (edge->fromNode != module.startNode)
        {
            // Need to traverse every edge in the path from root node to 
            // the initial edge's fromNode. To do so, we will do the following
            
            // go to the current node's previous edge
            int nodeInx = edge->fromNode;
            edge = &edges[nodes[nodeInx].previousEdge];

            // compute the product
            float _lambda = edge->radiiRatio;
            prod *= (_lambda * _lambda);
        }
        sum += l * prod * (1 + lambda + lambda * lambda);
    }
    return 1 / sqrt(sum);
}

__device__ float radiiUpdateRootNode(Node* nodes, Edge* edges, Module& module, float deltaMass)
{
    if (module.mass + deltaMass < FLT_EPSILON) return 0.f;
    return sqrt(3 / (M_PI * rho)) * radiiModuleConstant(nodes, edges, module) * sqrt(module.mass + deltaMass);
}

__device__ float radiiUpdateNode(Node* nodes, Edge* edges, Module& module, int nodeInx, float rootRadius)
{

    if (rootRadius < FLT_EPSILON) return 0.f;
    int currNodeInx = nodeInx;
    Edge* edge; // will be updated
    float prod = 1;
    do
    {
        // Need to traverse every edge in the path from root node to 
        // the node given. To do so, we will do the following

        // go to the current node's previous edge
        Node& node = nodes[currNodeInx];
        edge = &edges[node.previousEdge];
        currNodeInx = edge->fromNode;

        // compute the product
        prod *= edge->radiiRatio;
    } while (currNodeInx != module.startNode);
    return prod * rootRadius;
}

__device__ float rateOfTemperatureChange(float T, float T_M, float T_diff, float W, float A_M, float V_M)
{
    float b = (1 - W) * b_dry + W * b_wet; 
    float diffusion = alpha_M * T_diff; // Adjacent module diffusion  see eq. (30)
    float temp_diff = b * (T - T_M);

    float changeOfEnergy = 0;
    if (T_M > 150) // start of combustion
        changeOfEnergy = (c_bar * A_M * powf(T_M - T_sat, 3)) / (V_M * rho * c_M);
//# if __CUDA_ARCH__>=200
//    if (diffusion + temp_diff - changeOfEnergy != 0.f) {
//        printf("diffusion = %f, temp_diff = %f, changeOfEnergy = %f, T_M = %f, T_ENV = %f, alpha_M = %f\n", diffusion, temp_diff, changeOfEnergy, T_M, T, alpha_M);
//    }
//#endif 

//# if __CUDA_ARCH__>=200
//    if (diffusion + temp_diff - changeOfEnergy != 0.f) {
//        printf("diffusion = %f, temp_diff = %f, changeOfEnergy = %f, T_M = %f, T_ENV = %f\n", diffusion, temp_diff, changeOfEnergy, T_M, T);
//    }
//#endif

    return diffusion + temp_diff - changeOfEnergy;
}

__device__ float rateOfWaterChange(float changeInMass)
{
    return c_WM * changeInMass;
}

// Deprecated
__device__ float getDeltaMassOfModuleAtPoint(Module& module, glm::vec3 x, float dx)
{
    return -1;
    //return (1 - glm::distance(x, module.centerOfMass) / dx) * module.deltaM;
}

// Deprecated
__device__ float getWaterOfModuleAtPoint(Module& module, glm::vec3 x, float dx)
{
    return -1;
    /*return (1 - glm::distance(x, module.centerOfMass) / dx) * module.waterContent;*/
}

// Deprecated
__device__ float checkModuleIntersection(Module& module, glm::vec3 pos)
{
    /*bool intersects = true;
    for (int i = 0; i < 3; i++)
    {
        if (pos[i] < module.boundingMin[i] || pos[i] > module.boundingMax[i])
        {
            intersects = false;
            break;
        }
    }
    return intersects;*/
}

// TODO: add prototype to header file
__device__ float getModuleTemperatureLaplacian(Module* modules, ModuleEdge* moduleEdges, int moduleInx)
{
    Module& module = modules[moduleInx];
    float lap = 0.f;
    int sum = 0;

    // If module has no children or parent, return
    if ((module.startModule < 0 || module.endModule < 0) && module.parentModule < 0)
        return 0.f; 


    if (module.startModule >= 0 && module.endModule >= 0)
    {
        // Go through all the children of the module
        for (int i = module.startModule; i <= module.endModule; i++)
        {
            Module& adj = modules[moduleEdges[i].moduleInx];
            if (adj.culled) continue;
            float dist = glm::distance(module.centerOfMass, adj.centerOfMass);
            lap += (adj.temperature - module.temperature) / (dist * dist);
            //# if __CUDA_ARCH__>=200
            //if (lap != 0.f) {
            //    printf("%f\n", modules[3581].temperature);
            //    printf("adj[%d] = %f, module[%d] = %f\n", moduleEdges[i].moduleInx, adj.temperature, moduleInx, module.temperature);
            //}
            //#endif
            sum++;
        }
    }

    if (module.parentModule >= 0)
    {
        // Go through the module's parent
        Module& adj = modules[module.parentModule];
        if (!adj.culled)
        {
            float dist = glm::distance(module.centerOfMass, adj.centerOfMass);
            lap += (adj.temperature - module.temperature) / (dist * dist);
            sum++;
        }
    }
    return lap * lap_constant / sum;
}

/**********
* Kernels *
***********/

__global__ void kernInitModules(int N, Node* nodes, Edge* edges, Module* modules)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) return;

    Module& module = modules[index];

    if (module.startEdge < 0 || module.lastEdge < 0) {
        module.culled = true;
        return;
    }

    float area = 0.f;
    float mass = 0.f;
    for (int i = module.startEdge; i <= module.lastEdge; i++)
    {
        // for every edge in the module
        Edge& edge = edges[i];
        Node& fromNode = nodes[edge.fromNode];
        float r0 = fromNode.radius;
        float r1 = edge.radiiRatio * r0;
        float l = edge.length;
        area += getArea(r0, r1, l);
        float volume = getVolume(r0, r1, l);
        mass += volume * rho; // mass = density * volume 
    }

    glm::vec3 minPos{FLT_MAX}, maxPos{FLT_MIN};
    for (int i = module.startNode; i <= module.lastNode; i++)
    {
        // for every node in the module
        Node& node = nodes[i];
        glm::vec3 pos = node.position;
        for (int j = 0; j < 3; j++)
        {
            if (pos[j] < minPos[j])
                minPos[j] = pos[j];
            if (pos[j] > maxPos[j])
                maxPos[j] = pos[j];
        }
    }

    module.mass = mass;
    module.startArea = area;
    module.moduleConstant = radiiModuleConstant(nodes, edges, module);
    module.boundingMin = minPos;
    module.boundingMax = maxPos;

    module.deltaM = 0.f;
    module.temperature = T_amb;

    // primitive combustion
    /*if (index < 4000)
        module.temperature = 300;*/

    //module.waterContent = 0.f;
}

__global__ void kernInitIndices(int N, int* indices)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) return;

    indices[index] = index;
}

__global__ void kernModuleCombustion(float DT, int N, int* moduleIndices, int3 gridCount, float blockSize, Node* nodes, Edge* edges, Module* modules, ModuleEdge* moduleEdges, float* gridTemp)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) return;

    int moduleIndex = moduleIndices[index];

    if (moduleIndex == -1) return;

    /** for each module in the forest */
    Module& module = modules[moduleIndex];
    Node& rootNode = nodes[module.startNode];

    // Module needs to be culled
    if (module.mass < MASS_EPSILON || module.startEdge < 0 || module.lastEdge < 0 || module.culled) return;

    /** 1. update the mass */
    // calculate the current state of the module
    float mass = 0.f;
    float area = 0;
    float temp = module.temperature;

    for (int i = module.startEdge; i <= module.lastEdge; i++)
    {
        // for every branch in the module
        Edge& edge = edges[i];
        Node& fromNode = nodes[edge.fromNode];

        float r0 = fromNode.radius;

        float r1 = edge.radiiRatio * r0;
        float l = edge.length;

        float volume = getVolume(r0, r1, l);
        mass += volume * rho; // mass = density * volume 
        area += getArea(r0, r1, l);
    }

    // compute the change in mass
    float H0 = rootNode.startRadius;
    float A0 = module.startArea;
    float H = heightOfPyrolyzingFront(H0, A0, mass);
    float frontArea = getFrontArea(A0, H0, H);
    float windSpeed = 0; // TODO: implement
    float deltaM = glm::clamp(rateOfMassChange(mass, H0, H, A0, temp, frontArea, windSpeed), -MAX_DELTA_M, 0.f);
    //float deltaM = rateOfMassChange(mass, H0, H, A0, temp, frontArea, windSpeed);
    //if (deltaM != deltaM) deltaM = -0.001f;
    /*if (moduleIndex < 12500 && module.temperature > 150)
        deltaM = -0.0005f;
    else
        deltaM = 0.f;*/

    module.mass += deltaM;
    module.deltaM = deltaM;

    /** Perform radii update */
    // update the root's radius
    float rootRadius = radiiUpdateRootNode(nodes, edges, module, deltaM);
    rootNode.radius = rootRadius;

    for (int i = module.startNode + 1; i <= module.lastNode; i++)
    {
        // update the radius of each branch in the module
        Node& node = nodes[i];
        float newRadius = radiiUpdateNode(nodes, edges, module, i, rootRadius);
        node.radius = newRadius;
    }

//# if __CUDA_ARCH__>=200
//
//    if (modules[moduleIndex].temperature != 15.0f || moduleIndex == 3581) {
//        printf("module[%d]: %f\n", moduleIndex, modules[moduleIndex].temperature);
//    }
//#endif

    /** 3. Update temperature */
    float T_env = gridTemp[module.gridCell];
    float T_diff = getModuleTemperatureLaplacian(modules, moduleEdges, moduleIndex);
    float T_M = module.temperature;
    float W = 0; // TODO: get the water content
    float A_M = area; // lateral surface area 
    float V_M = module.mass / rho;

//# if __CUDA_ARCH__>=200
//
//    if (modules[moduleIndex].temperature != 15.0f || moduleIndex == 3581) {
//        printf("module[%d]: %f\n", moduleIndex, modules[moduleIndex].temperature);
//    }
//#endif

//# if __CUDA_ARCH__>=200
//    if (T_diff != 0.f) {
//        printf("Module = %d, temp_diff = %f\n", moduleIndex, T_diff);
//    }
//#endif


    //float deltaT = glm::clamp(rateOfTemperatureChange(T_env, T_M, T_diff, W, A_M, V_M), -MAX_DELTA_T, MAX_DELTA_T);
    float deltaT = rateOfTemperatureChange(T_env, T_M, T_diff, W, A_M, V_M);

    module.temperature += deltaT;

    /** 4. Update released water content */
    /*float deltaW = rateOfWaterChange(deltaM);
    module.waterContent += deltaW;*/
}

// TODO: these functions are duplicated in advection.h/.cpp
__device__ int m_idxClip(int idx, int idxMax) {
    return idx > (idxMax - 1) ? (idxMax - 1) : (idx < 0 ? 0 : idx);
}
__device__ int m_flatten(int col, int row, int z, int width, int height, int depth) {
    return m_idxClip(col, width) + m_idxClip(row, height) * width + m_idxClip(z, depth) * width * height;
}
__global__ void kernComputeChangeInMass(int3 gridCount, Module* modules, GridCell* gridCells, GridModuleAdj* gridModuleAdjs, float* gridOfMass)
{
    const int k_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int k_y = threadIdx.y + blockDim.y * blockIdx.y;
    const int k_z = threadIdx.z + blockDim.z * blockIdx.z;
    if ((k_x >= gridCount.x) || (k_y >= gridCount.y) || (k_z >= gridCount.z)) return;
    const int k = m_flatten(k_x, k_y, k_z, gridCount.x, gridCount.y, gridCount.z);

    GridCell& gridCell = gridCells[k];
    if (gridCell.startModule < 0 || gridCell.endModule < 0)
        return;

    float deltaM = 0.f;
    for (int i = gridCell.startModule; i <= gridCell.endModule; i++)
    {
        int index = gridModuleAdjs[i].moduleInx;
        if (modules[i].deltaM < -MAX_DELTA_M || modules[i].deltaM > 0.f) {
            continue;
        }
        deltaM += modules[index].deltaM;
    }
//    # if __CUDA_ARCH__>=200
//    if (gridCell.endModule - gridCell.startModule > 100) {
//        printf("start = %d, end = %d\n", gridCell.startModule, gridCell.endModule);
//    }
//#endif

    gridOfMass[k] = deltaM;
}

__device__ float getGridCell(Module& module, int3 gridCount, float blockSize)
{
    // Convert center of mass to grid-space coordinates // e.g. (-10,10) to (0, 20)
    glm::vec3 com = module.centerOfMass;
    com.x += floor(gridCount.x * blockSize / 2);
    com.y += floor(gridCount.y * blockSize / 2);
    com.z += floor(gridCount.z * blockSize / 2);

    // get the grid at this location
    for (int i = 0; i < 3; i++)
        com[i] = blockSize * round(com[i] / blockSize);
    int inx = m_flatten(com.x, com.y, com.z, gridCount.x, gridCount.y, gridCount.z);

    return inx;
}

__device__ float getEnvironmentTempAtModule(Module& module, float* temp, int3 gridCount, float blockSize)
{
    int inx = getGridCell(module, gridCount, blockSize);

    return temp[inx];
}

__global__ void kernCullModules1(int N, int* moduleIndices, Module* modules, ModuleEdge* moduleEdges, Node* nodes, Edge* edges)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) return;
    
    int moduleInx = moduleIndices[index];
    if (moduleInx < 0) return;

    Module& module = modules[moduleInx];

    // check if module needs to be culled
    if (module.mass < MASS_EPSILON || module.startEdge < 0 || module.lastEdge < 0)
    {
        module.culled = true;
    }
    # if __CUDA_ARCH__>=200
    if (moduleInx == 5287 && module.culled) {
        printf("culled 5287\n");
    }
#endif 
}

__global__ void kernCullModules2(int N, int* moduleIndices, Module* modules, ModuleEdge* moduleEdges, Node* nodes, Edge* edges)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) return;

    int moduleInx = moduleIndices[index];
    if (moduleInx < 0) return;

    // check if culled or one of its parent modules has been culled
    Module& module = modules[moduleInx];
    Module* currModule = &modules[moduleInx];
    while (currModule->parentModule >= 0 && !module.culled)
    {
        currModule = &modules[currModule->parentModule];
        if (currModule->mass < MASS_EPSILON)
        {
            module.culled = true;
            break;
        }
    }
    if (module.culled)
    {
        moduleIndices[index] = -1; // cull the module

        // reset module values
        module.deltaM = 0.f;

        // for every edge in the module, cull it so it isn't rendered
        if (module.startEdge < 0 || module.lastEdge < 0) return;
        for (int i = module.startEdge; i <= module.lastEdge; i++)
        {
            Edge& edge = edges[i];
            edge.culled = true;
            if (edge.fromNode >= 0 && edge.toNode >= 0)
            {
                nodes[edge.fromNode].radius = 0.f;
                nodes[edge.toNode].radius = 0.f;
            }
        }
    }
}